#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

typedef struct {
	int w, h;
	uchar4 *data;
} image;


// текстурная ссылка <тип элементов, размерность, режим нормализации>
texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(
	uchar4 *out, int w, int h, 
	float *fmatrix, int msize
) {
	int idx     = blockDim.x * blockIdx.x + threadIdx.x;
	int idy     = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;

	for (y = idy; y < h; y += offsety)
		for (x = idx; x < w; x += offsetx) {
			// uchar4 newcol;
			uchar4 p;

			// p  = tex2D(tex, x, y);
			
			// newcol.x = p.x * 0.15;
			// newcol.y = p.y * 0.7;
			// newcol.z = p.z * 0.15;
			
			// out[y * w + x] = newcol;

			int lx, ly;
			int mcenter = msize / 2;
			double col[] = {0, 0, 0};
			for (lx = 0; lx < msize; ++lx)
				for (ly = 0; ly < msize; ++ly) {
					p = tex2D(tex, x + lx - mcenter, y + ly - mcenter);
					col[0] += (int)p.x * fmatrix[ly * msize + lx];
					col[1] += (int)p.y * fmatrix[ly * msize + lx];
					col[2] += (int)p.z * fmatrix[ly * msize + lx];
				}

			for (int i = 0; i < 3; ++i)
				//col[i] /= 2;
				if (col[i] > 255)
					col[i] = 255;

			out[y * w + x] = make_uchar4(col[0], col[1], col[2], 0);
		}
}

image img_read(char *fname) {
	unsigned char *tdata;
	image img;
	int t;

	fprintf(stderr, "loading image %s\n", fname);
	tdata = stbi_load(fname, &img.w, &img.h, &t, 4);
	fprintf(stderr, "size: %dx%d\n", img.w, img.h);

	img.data = (uchar4 *)malloc(sizeof(uchar4) * img.w * img.h);
	memcpy(img.data, tdata, sizeof(uchar4) * img.w * img.h);

	free(tdata);
	fprintf(stderr, "finished loading\n");

	return img;
}


int img_write(char *fname, image img) {
	int err = 0;
	
	// if (str_ends_with(fname, ".jpg"))
	fprintf(stderr, "Writing data to %s\n", fname);
	// printf("%d%d%d\n", img.data[0].x, img.data[0].y, img.data[0].z);
	stbi_write_jpg(fname, img.w, img.h, 4, img.data, 100);
	// stbi_write_bmp(fname, img.w, img.h, 4, img.data);

	return err;
}

int main(int argc, char **argv) {
	int     err = 0;
	char   *inm = NULL;
	char   *onm = NULL;
	char    t1[100];
	char    t2[100];
	// uchar4 *data;
	hipEvent_t start, end;
	image input;

	if (argc == 1) {
		scanf("%s", t1); 
		scanf("%s", t2);
			
		inm = t1;
		onm = t2;
	}
	else {
		inm = argv[1];
		onm = argv[2];
	}

	if (!err) {
		// finp = fopen(inm, "rb");
		// fout = fopen(onm, "wb");
		
		fprintf(stderr, "inname: %s\n", inm);
		fprintf(stderr, "otname: %s\n", onm);
	}
	
	input = img_read(inm);
	
	fprintf(stderr, "%dx%d\n", input.w, input.h);
	
	// data = (uchar4 *)malloc(sizeof(uchar4) * input.w * input.h);
	
	if (input.data) {
		hipArray *arr;
		
		// fprintf(stderr, "Reading data\n");
		// fread(data, sizeof(uchar4), w * h, finp);

		// fprintf(stderr, "Reading ok\n");

		// Подготовка данных для текстуры
		hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
		CSC(hipMallocArray(&arr, &ch, input.w, input.h));
		
		if (arr) {
			float  *hst_fmatrix;
			float  *dev_fmatrix;
			int     fmsize;

			scanf("%d", &fmsize);
			hst_fmatrix = (float *)malloc(sizeof(float) * fmsize * fmsize);
			for (int i = 0; i < fmsize * fmsize; ++i)
				scanf("%g", hst_fmatrix + i);
			CSC(hipMalloc(&dev_fmatrix, fmsize * fmsize * sizeof(float)));
			CSC(hipMemcpy( dev_fmatrix, hst_fmatrix, sizeof(float) * fmsize * fmsize, hipMemcpyHostToDevice));
				
			uchar4 *dev_out;
				
			CSC(hipMemcpyToArray(arr, 0, 0, input.data, sizeof(uchar4) * input.w * input.h, hipMemcpyHostToDevice));

			// Подготовка текстурной ссылки, настройка интерфейса работы с данными
			tex.addressMode[0] = hipAddressModeClamp;	// Политика обработки выхода за границы по каждому измерению
			tex.addressMode[1] = hipAddressModeClamp;
			tex.channelDesc = ch;
			//tex.filterMode  = hipFilterModePoint; // original 
			//tex.normalized  = false;
			tex.filterMode  = hipFilterModePoint;
			tex.normalized  = false;

			// Связываем интерфейс с данными
			CSC(hipBindTextureToArray(tex, arr, ch));

			CSC(hipMalloc(&dev_out, sizeof(uchar4) * input.w * input.h));

			fprintf(stderr, "Launching kernel\n");
			CSC(hipEventCreate(&start));
			CSC(hipEventCreate(&end  ));
			CSC(hipEventRecord( start));
			//~ fprintf(stderr, "Launching kernel\n");
			//kernel<<<dim3(GRID_SIZE, GRID_SIZE), dim3(BLOCK_SIZE, BLOCK_SIZE)>>>(dev_out, w, h, dev_fmatrix, fmsize);
			int gridsize = 16;
			int blocksize = 32;
			
			kernel<<<dim3(gridsize, gridsize), dim3(blocksize, blocksize)>>>(dev_out, input.w, input.h, dev_fmatrix, fmsize);
			
			CSC(hipGetLastError());
			CSC(hipEventRecord     (end));
			CSC(hipEventSynchronize(end));
			float t;
			CSC(hipEventElapsedTime(&t, start, end));
			CSC(hipEventDestroy(start));
			CSC(hipEventDestroy(end));
			fprintf(stderr, "Finished kernel\n");

			fprintf(stderr, "time = %f\n", t);
			//~ fprintf(stderr, "Kernel finished\n");

			CSC(hipMemcpy(input.data, dev_out, sizeof(uchar4) * input.w * input.h, hipMemcpyDeviceToHost));
			fprintf(stderr, "Copying back finished\n");

			// Отвязываем данные от текстурной ссылки
			CSC(hipUnbindTexture(tex));
			fprintf(stderr, "Unbind finished\n");

			//~ CSC(hipFreeArray(arr));
			//~ CSC(hipFree(dev_out));
		}
		else
			err = 97;
	}
	else
		err = 96;

	// fclose(finp);
	
	// if (fout) {
	// fprintf(stderr, "Writing data\n");

	// fwrite(&w,   sizeof(int), 1, fout);
	// fwrite(&h,   sizeof(int), 1, fout);
	// fwrite(data, sizeof(uchar4), w * h, fout);
	
	// fclose(fout);
	//~ free  (data);
	img_write(onm, input);
	
	// fprintf(stderr, "Data written\n");
	// }
	// else
	// 	err = 98;
		
	if (err)
		fprintf(stderr, "E: error with code %d has occured!\n", err);
	
	return err;
}
