
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)                      \
do {                                \
    hipError_t res = call;            \
    if (res != hipSuccess) {        \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(res));        \
        exit(0);                    \
    }                                \
} while(0)

__global__ void kernel(int *arr1, int n) { //global значит ф-ию можно вызвать из хоста, несмотря на то что выполняться она будет на девайсе
    int idx = threadIdx.x + blockIdx.x * blockDim.x; //магия связанная с кол-м тредов и распределением чиселок по ним
    int offset = blockDim.x * gridDim.x;//на сколько раз перемещаемся в каждом из шагов вцикле
    
    while(idx < n/2) {
		int tmp = arr1[idx];
        arr1[idx] = arr1[n - idx - 1];
        arr1[n - idx - 1] = tmp;
        idx += offset;
    }
}

__global__ void kernelByt(int *arr1, int n) { //global значит ф-ию можно вызвать из хоста, несмотря на то что выполняться она будет на девайсе
    int idx = threadIdx.x + blockIdx.x * blockDim.x; //магия связанная с кол-м тредов и распределением чиселок по ним
    int offset = blockDim.x * gridDim.x;//на сколько раз перемещаемся в каждом из шагов вцикле
    
    while(idx < n) {
		arr1[idx] = ~arr1[idx];
        idx += offset;
    }
}


int main(){
	int *arrHost = (int *)malloc(512*sizeof(int));
	int *arrDev;
	int i;
	
	for (i = 0; i < 512; i++)
		arrHost[i] = i + 1;
	
    hipEvent_t before, after;
    CSC(hipEventCreate(&before));
    CSC(hipEventCreate(&after));

	CSC(hipMalloc(&arrDev, 512*sizeof(int)));
	CSC(hipMemcpy(arrDev, arrHost, 512*sizeof(int), hipMemcpyHostToDevice));
	
    CSC(hipEventRecord(before));
	//~ kernelByt<<<256, 256>>>(arrDev, 512);  //кол-во threadов
	kernel<<<256, 256>>>(arrDev, 512);
	
    CSC(hipGetLastError());
    CSC(hipEventRecord(after));

    CSC(hipEventSynchronize(after));
    float t;
    CSC(hipEventElapsedTime(&t, before, after));
    CSC(hipEventDestroy(before));
    CSC(hipEventDestroy(after));

    printf("time = %f\n", t); //выводим посчитанную разницу во времени
    
    CSC(hipMemcpy(arrHost, arrDev, 512*sizeof(int), hipMemcpyDeviceToHost));
    
    for (i = 0; i < 512; i++){
		printf("%d ", arrHost[i]);
	}	
	printf("\n");
	CSC(hipFree(arrDev));
	free(arrHost);
	return 0;
}
