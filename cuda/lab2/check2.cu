#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// #include <stdint.h>

typedef unsigned long long int uint64_t;

#define PI_CNT 16
#define BL_CNT  16
#define PER_THREAD 64 / PI_CNT

#define CSC(call)                      \
do {                                \
    hipError_t res = call;            \
    if (res != hipSuccess) {        \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
            __FILE__, __LINE__, hipGetErrorString(res));        \
        exit(0);                    \
    }                                \
} while(0)

__global__ void kernel(uint64_t number, uint64_t *ipowers, uint64_t *result, uint64_t *pokasa) {
    __shared__ uint64_t power_array[64]; 
    
    // вычисляется индекс элемента который будет обрабатываться
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 

    //данный массив в начале заполняется степенями, которые были вычислены "по-быстрому"
    power_array[idx] = ipowers[idx / (64/PI_CNT)]; 

    //показатель степени, которая сейчас лежит в текущем эл-те массива
    int pokasatel_stepeni = pokasa[idx / (64/PI_CNT)]; 

    while (pokasatel_stepeni < idx) { //доводим степень до нужной
        power_array[idx] *= 2;
        pokasatel_stepeni++;
    }

    __syncthreads();

    result[idx] = power_array[idx];
}

int main(void) {
    uint64_t  initial_powers[PI_CNT];
    uint64_t *result = (uint64_t *)malloc(64 * sizeof(uint64_t));
    uint64_t  pokasa[PI_CNT];
    uint64_t step = pow(2, 64 / PI_CNT);
    uint64_t *ipowers_dev;
    uint64_t *result_dev;
    uint64_t *pokasa_dev; 
    uint64_t  number;

    hipEvent_t before, after;
    CSC(hipEventCreate(&before)); // инициализируем 2 события cuda
    CSC(hipEventCreate(&after));

    printf("Enter the number: ");
    scanf ("%llu", &number);

    initial_powers[0] = 1;
    pokasa        [0] = 0;
    for (int i = 1; i < PI_CNT; ++i) {
        initial_powers[i] = initial_powers[i - 1] * step;
        pokasa        [i] = pokasa        [i - 1] + 64 / PI_CNT;
    }

    CSC(hipMalloc(&ipowers_dev,                 PI_CNT * sizeof(uint64_t)));
    CSC(hipMalloc(&pokasa_dev,                  PI_CNT * sizeof(uint64_t)));
    CSC(hipMemcpy( ipowers_dev, initial_powers, PI_CNT * sizeof(uint64_t), hipMemcpyHostToDevice));
    CSC(hipMemcpy( pokasa_dev,  pokasa,         PI_CNT * sizeof(uint64_t), hipMemcpyHostToDevice));

    CSC(hipMalloc(&result_dev, 64 * sizeof(uint64_t)));

    CSC(hipEventRecord(before)); // сохраняем текущее время начала работы ядра
    kernel<<<BL_CNT, 64 / BL_CNT>>>(number, ipowers_dev, result_dev, pokasa_dev);
    CSC(hipGetLastError());
    CSC(hipEventRecord(after)); // сохраняем время конца работы ядра

    CSC(hipEventSynchronize(after));
    float t;
    CSC(hipEventElapsedTime(&t, before, after)); // считаем время работы ядра
    CSC(hipEventDestroy(before));
    CSC(hipEventDestroy(after));
    printf("TIME = %g\n", t);

    CSC(hipMemcpy(result, result_dev, 64 * sizeof(uint64_t), hipMemcpyDeviceToHost));

    int tmp = -1;
    for (int i = 0; i < 64; ++i)
        if (result[i] == number)
            tmp = i;
    
    if (tmp > 0)
        printf("number %llu is %dth power of 2\n", number, tmp);
    else
        printf("Nope. it's no power of 2\n");

    return 0;
}