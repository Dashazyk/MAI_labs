#include "hip/hip_runtime.h"
#include <stdio.h>
#include "vect3.h"

__host__ __device__ double dot(vect3 a, vect3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

// prod
__host__ __device__ vect3 prod(vect3 a, vect3 b) {
	return {
		a.y * b.z - a.z * b.y, 
		a.z * b.x - a.x * b.z, 
		a.x * b.y - a.y * b.x
	};
}

__host__ __device__ vect3 norm(vect3 v) {
	double l = sqrt(dot(v, v));
	return {
		v.x / l, 
		v.y / l, 
		v.z / l
	};
}

__host__ __device__ vect3 diff(vect3 a, vect3 b) {
	return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__host__ __device__ vect3 add(vect3 a, vect3 b) {
	return {a.x + b.x, a.y + b.y, a.z + b.z};
}
__host__ __device__ uchar4 add(uchar4 a, uchar4 b) {
	return {
		(uchar)MIN((int)a.x + b.x, 255), 
		(uchar)MIN((int)a.y + b.y, 255),
		(uchar)MIN((int)a.z + b.z, 255),
		(uchar)MIN((int)a.w + b.w, 255)
	};
}

__host__ __device__ vect3 mult(vect3 a, vect3 b, vect3 c, vect3 v) {
	return {
		a.x * v.x + b.x * v.y + c.x * v.z,
		a.y * v.x + b.y * v.y + c.y * v.z,
		a.z * v.x + b.z * v.y + c.z * v.z
	};
}

__host__ __device__ vect3 scale(vect3 v, double m) {
	return {v.x * m, v.y * m, v.z * m};
}
__host__ __device__ uchar4 scale(uchar4 v, double m) {
	return {
		(uchar)(v.x * m), 
		(uchar)(v.y * m), 
		(uchar)(v.z * m), 
		(uchar)(v.w * m)
	};
}

__host__ __device__ double length(vect3 v) {
	//~ return sqrt(dot(v, v));
	return sqrt(v.x * v.x + v.y*v.y + v.z * v.z);
}

__host__ __device__ void print(char *nm, vect3 v) {
	printf("%s (%g; %g; %g)\n", nm, v.x, v.y, v.z);
}

__host__ __device__ double dbl2limits(double value, double down, double upper) {
	return (upper - down)/1.0 * value + down;
}

__host__ __device__ vect3 v2limits(vect3 value, vect3 down, vect3 upper) {
	return {
		dbl2limits(value.x, down.x, upper.x),
		dbl2limits(value.y, down.y, upper.y),
		dbl2limits(value.z, down.z, upper.z)
	};
}
