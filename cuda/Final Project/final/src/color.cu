#include "hip/hip_runtime.h"
#include "Triangle.h"
#include "color.h"

typedef unsigned char uchar;

__host__ __device__ uchar4 u4color(vect3 v) {
	return {
		(uchar)(v.x * 255),
		(uchar)(v.y * 255),
		(uchar)(v.z * 255),
		255
	};
}
