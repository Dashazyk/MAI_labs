#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "Triangle.h"
#include "color.h"


Triangle make_triangle(vect3 a, vect3 b, vect3 c, uchar4 color) {
	Triangle trig = {a, b, c, color};
	trig.normal = norm(prod(
		diff(trig.a, trig.c),
		diff(trig.a, trig.b)
	));
	return trig;
}

Triangle make_triangle(vect3 a, vect3 b, vect3 c, vect3 color) {
	return make_triangle(a, b, c, u4color(color));
}
