#include "hip/hip_runtime.h"
#include <stdio.h>
#include "fig.h"

Figure gen_floor(vect3 color, vect3 *points) {
	Figure f;

	f.triag_count = 2;
	f.triags = (Triangle *)malloc(sizeof(Triangle) * f.triag_count);
	
	f.triags[0] = make_triangle(points[1], points[0], points[2], color);
	f.triags[1] = make_triangle(points[0], points[1], points[3], color);

	return f;
}

Figure gen_icosa(vect3 color, vect3 pos, double rad) {
	Figure f;

	f.triag_count = 20;
	f.triags = (Triangle *)malloc(sizeof(Triangle) * f.triag_count);
	
    vect3 v[12];
	float X=.525731112119133606f;
	float Z=.850650808352039932f;
	float N=0.f;


	fprintf(stderr, "%s (%g) ", __FUNCTION__, rad);
	//~ print(pos);
	fprintf(stderr, "=======\n");

	v[0]  = {-X, N, Z};
	v[1]  = { X, N, Z};
	v[2]  = {-X, N,-Z};
	v[3]  = { X, N,-Z};
	v[4]  = { N, Z, X};
	v[5]  = { N, Z,-X};
	v[6]  = { N,-Z, X};
	v[7]  = { N,-Z,-X};
	v[8]  = { Z, X, N};
	v[9]  = {-Z, X, N};
	v[10] = { Z,-X, N};
	v[11] = {-Z,-X, N};
	
    for (int i = 0; i < 12; i++)
		v[i] = add(scale(v[i], rad), pos);
    
	f.triags[ 0] = make_triangle(v[ 0], v[ 4], v[ 1], color);
	f.triags[ 1] = make_triangle(v[ 0], v[ 9], v[ 4], color);
	f.triags[ 2] = make_triangle(v[ 9], v[ 5], v[ 4], color);
	f.triags[ 3] = make_triangle(v[ 4], v[ 5], v[ 8], color);
	f.triags[ 4] = make_triangle(v[ 4], v[ 8], v[ 1], color);
	f.triags[ 5] = make_triangle(v[ 8], v[10], v[ 1], color);
	f.triags[ 6] = make_triangle(v[ 8], v[ 3], v[10], color);
	f.triags[ 7] = make_triangle(v[ 5], v[ 3], v[ 8], color);
	f.triags[ 8] = make_triangle(v[ 5], v[ 2], v[ 3], color);
	f.triags[ 9] = make_triangle(v[ 2], v[ 7], v[ 3], color);
	f.triags[10] = make_triangle(v[ 7], v[10], v[ 3], color);
	f.triags[11] = make_triangle(v[ 7], v[ 6], v[10], color);
	f.triags[12] = make_triangle(v[ 7], v[11], v[ 6], color);
	f.triags[13] = make_triangle(v[11], v[ 0], v[ 6], color);
	f.triags[14] = make_triangle(v[ 0], v[ 1], v[ 6], color);
	f.triags[15] = make_triangle(v[ 6], v[ 1], v[10], color);
	f.triags[16] = make_triangle(v[ 9], v[ 0], v[11], color);
	f.triags[17] = make_triangle(v[ 9], v[11], v[ 2], color);
	f.triags[18] = make_triangle(v[ 9], v[ 2], v[ 5], color);
	f.triags[19] = make_triangle(v[ 7], v[ 2], v[11], color); 

	return f;
}

Figure gen_hex(vect3 color, vect3 pos, double rad) {
	Figure f;
	
	f.triag_count = 16;
	f.triags = (Triangle *)malloc(sizeof(Triangle) * f.triag_count);
	//~ memset(f.triags, 0, sizeof(Triangle) * f.triag_count);

	fprintf(stderr, "%s (%g)\n", __FUNCTION__, rad);
	//~ print(pos);

	vect3 ft = { 0,  1,  0};
	vect3 bk = { 0, -1,  0};
	vect3 tp = { 0,  0,  1};
	vect3 bm = { 0,  0, -1};
	vect3 lt = {-1,  0,  0};
	vect3 rt = { 1,  0,  0};

	//~ ft = add(scale(ft, rad), pos);
	//~ bk = add(scale(bk, rad), pos);
	//~ tp = add(scale(tp, rad), pos);
	//~ bm = add(scale(bm, rad), pos);
	//~ lt = add(scale(lt, rad), pos);
	//~ rt = add(scale(rt, rad), pos);

	vect3 lt_ft_tp = add(add(lt, ft), tp);
	vect3 lt_ft_bm = add(add(lt, ft), bm);
	
	vect3 lt_bk_tp = add(add(lt, bk), tp);
	vect3 lt_bk_bm = add(add(lt, bk), bm);
	
	vect3 rt_ft_tp = add(add(rt, ft), tp);
	vect3 rt_ft_bm = add(add(rt, ft), bm);
	
	vect3 rt_bk_tp = add(add(rt, bk), tp);
	vect3 rt_bk_bm = add(add(rt, bk), bm);

	rad /= sqrt(3);
	lt_ft_tp = add(scale(lt_ft_tp, rad), pos);
	lt_ft_bm = add(scale(lt_ft_bm, rad), pos);
	
	lt_bk_tp = add(scale(lt_bk_tp, rad), pos);
	lt_bk_bm = add(scale(lt_bk_bm, rad), pos);
	
	rt_ft_tp = add(scale(rt_ft_tp, rad), pos);
	rt_ft_bm = add(scale(rt_ft_bm, rad), pos);
	
	rt_bk_tp = add(scale(rt_bk_tp, rad), pos);
	rt_bk_bm = add(scale(rt_bk_bm, rad), pos);

	f.triags[ 0] = make_triangle(lt_ft_bm, lt_ft_tp, rt_ft_tp, color);
	f.triags[ 1] = make_triangle(lt_ft_bm, rt_ft_bm, rt_ft_tp, color);
	
	f.triags[ 2] = make_triangle(lt_ft_tp, lt_bk_tp, rt_ft_tp, color);
	f.triags[ 3] = make_triangle(rt_bk_tp, lt_bk_tp, rt_ft_tp, color);
	
	f.triags[ 4] = make_triangle(lt_ft_bm, lt_bk_bm, rt_ft_bm, color);
	f.triags[ 5] = make_triangle(rt_bk_bm, lt_bk_bm, rt_ft_bm, color);

	f.triags[ 6] = make_triangle(lt_bk_bm, lt_bk_tp, rt_bk_tp, color);
	f.triags[ 7] = make_triangle(lt_bk_bm, rt_bk_bm, rt_bk_tp, color);
	
	f.triags[ 8] = make_triangle(lt_ft_tp, lt_ft_bm, lt_bk_tp, color);
	f.triags[ 9] = make_triangle(lt_bk_tp, lt_bk_bm, lt_ft_bm, color);
	
	f.triags[10] = make_triangle(rt_ft_tp, rt_ft_bm, rt_bk_tp, color);
	f.triags[11] = make_triangle(rt_bk_tp, rt_bk_bm, rt_ft_bm, color);

	//~ print(tp);
	//~ print(bm);
	
	//~ print(lt_ft_bm);
	//~ print(rt_bk_tp);
	fprintf(stderr, "=======\n");
	//~ f.triags[] = make_triangle(, color);
	//~ f.triags[] = make_triangle(, color);
	//~ f.triags[] = make_triangle(, color);
	//~ f.triags[] = make_triangle(, color);
	//~ f.triags[] = make_triangle(, color);
	//~ f.triags[] = make_triangle(, color);

	return f;
}

Figure gen_dodec(vect3 color, vect3 pos, double rad) {
	Figure f;
	f.triag_count = 37;
	f.triags = (Triangle *)malloc(sizeof(Triangle) * f.triag_count);

	fprintf(stderr, "%s (%g) ", __FUNCTION__, rad);
	//~ print(pos);
	fprintf(stderr, "=======\n");
	
	float u = 2 / (sqrtf(5) + 1), p = (sqrtf(5) + 1)/2;

    //std::cout << "p - " << p << "\n";
    //std::cout << "u - " << u << "\n";

    vect3 v[20] = {
		vect3{-u, 0, p},   vect3{u, 0, p},
		vect3{-1, 1, 1},   vect3{1, 1, 1},
		vect3{1, -1, 1},   vect3{-1, -1, 1},
		vect3{0, -p, u},   vect3{0, p, u},
		vect3{-p, -u, 0},  vect3{-p, u, 0},
		vect3{p, u, 0},    vect3{p, -u, 0},
		vect3{0, -p, -u},  vect3{0, p, -u},
		vect3{1, 1, -1},   vect3{1, -1, -1},
		vect3{-1, -1, -1}, vect3{-1, 1, -1},
		vect3{u, 0, -p},   vect3{-u, 0, -p}
    };

    for (int i = 0; i < 20; i++) {
        v[i].x /= sqrt(3);
        v[i].y /= sqrt(3);
        v[i].z /= sqrt(3);
    }
    
    /*std::cout << "v[11] - " << v[11].x << "\n";
    std::cout << "v[12] - " << v[12].x << "\n";
    std::cout << "v[13] - " << v[13].x << "\n";
    std::cout << "v[14] - " << v[14].x << "\n";
    std::cout << "v[15] - " << v[15].x << "\n";
    std::cout << "v[16] - " << v[16].x << "\n";
    std::cout << "v[17] - " << v[17].x << "\n";
    std::cout << "v[18] - " << v[18].x << "\n";
    std::cout << "v[19] - " << v[19].x << "\n";*/

    f.triags[ 0] = make_triangle(v[4], v[0], v[6], color);
    f.triags[ 1] = make_triangle(v[0], v[5], v[6], color);
    f.triags[ 2] = make_triangle(v[0], v[4], v[1], color);
    f.triags[ 3] = make_triangle(v[0], v[3], v[7], color);
    f.triags[ 4] = make_triangle(v[2], v[0], v[7], color);
    f.triags[ 5] = make_triangle(v[0], v[1], v[3], color);
    
    f.triags[ 6] = make_triangle(v[10], v[1], v[11], color);
    f.triags[ 7] = make_triangle(v[3], v[1], v[10], color);
    f.triags[ 8] = make_triangle(v[1], v[4], v[11], color);
    f.triags[ 9] = make_triangle(v[5], v[0], v[8], color);
    f.triags[10] = make_triangle(v[0], v[2], v[9], color);
    f.triags[11] = make_triangle(v[8], v[0], v[9], color);
    
    f.triags[12] = make_triangle(v[5], v[8], v[16], color);
    f.triags[13] = make_triangle(v[6], v[5], v[12], color);
    f.triags[14] = make_triangle(v[12], v[5], v[16], color);
    f.triags[15] = make_triangle(v[4], v[12], v[15], color);
    f.triags[16] = make_triangle(v[4], v[6], v[12], color);
    f.triags[17] = make_triangle(v[11], v[4], v[15], color);
    
    f.triags[18] = make_triangle(v[2], v[13], v[17], color);
    f.triags[19] = make_triangle(v[2], v[7], v[13], color);
    f.triags[20] = make_triangle(v[9], v[2], v[17], color);
    f.triags[21] = make_triangle(v[13], v[3], v[14], color);
    f.triags[22] = make_triangle(v[7], v[3], v[13], color);
    f.triags[23] = make_triangle(v[3], v[10], v[14], color);
    
    f.triags[24] = make_triangle(v[8], v[17], v[19], color);
    f.triags[25] = make_triangle(v[16], v[8], v[19], color);
    f.triags[26] = make_triangle(v[8], v[9], v[17], color);
    f.triags[27] = make_triangle(v[14], v[11], v[18], color);
    f.triags[28] = make_triangle(v[11], v[15], v[18], color);
    f.triags[29] = make_triangle(v[10], v[11], v[14], color);
    
    f.triags[31] = make_triangle(v[12], v[19], v[18], color);
    f.triags[32] = make_triangle(v[15], v[12], v[18], color);
    f.triags[33] = make_triangle(v[12], v[16], v[19], color);
    f.triags[34] = make_triangle(v[19], v[13], v[18], color);
    f.triags[35] = make_triangle(v[17], v[13], v[19], color);
    f.triags[36] = make_triangle(v[13], v[14], v[18], color);

    //std::cout << "tr[46] - " << triags[46].a.x << "\n";
    //std::cout << "radius - " << Figure.radius << "\n";
    for (int i = 0; i < 37; i++) {
        f.triags[i].a = scale(f.triags[i].a, rad);
        f.triags[i].b = scale(f.triags[i].b, rad);
        f.triags[i].c = scale(f.triags[i].c, rad);
        
        //if(i == 12 || i == 13 || i == 14) std::cout << "trg.a.x * radius - " << triags[i].a.x << "\n";

        f.triags[i].a = add(f.triags[i].a, pos);
        f.triags[i].b = add(f.triags[i].b, pos);
        f.triags[i].c = add(f.triags[i].c, pos);
        
        //if(i == 12 || i == 13 || i == 14) std::cout << "trg.a.x + cntr - " << triags[i].a.x << "\n";
        
        //std::cout << triags[i].a.x << "\n";
    }

	return f;
}
