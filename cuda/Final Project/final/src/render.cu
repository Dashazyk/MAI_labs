#include "hip/hip_runtime.h"
#include <omp.h>
#include "render.h"

struct ray_intersection {
	double dst_min;
	int index;
	vect3 intersection;
};

__host__ __device__ ray_intersection ray(Triangle *triags, int trig_cnt, vect3 pos, vect3 dir) {
	int k, k_min = -1;
	double ts_min = 100;

	dir = norm(dir);

	ray_intersection result;
	
	for (k = 0; k < trig_cnt; k++) {
		vect3 side1 = diff(triags[k].b, triags[k].a); 
		vect3 side2 = diff(triags[k].c, triags[k].a); 
		vect3 p = prod(dir, side2); 
		
		double div = dot(p, side1);
		if (fabs(div) >= 1e-10) {
			vect3 t = diff(pos, triags[k].a);
			double u = dot(p, t) / div;
			if (!(u < 0.0 || u > 1.0)) {
				vect3  q = prod(t, side1);
				double v = dot(q, dir) / div;
				
				if (!(v < 0.0 || v + u > 1.0)) {
					double ts = dot(q, side2) / div;
					if (ts >= 0.0) {
						if (k_min == -1 || ts < ts_min) {
							k_min = k;
							ts_min = ts;
						}
					}
				}
			}
		}
	}

	result.index = k_min;
	if (k_min != -1) {
		result.dst_min   = ts_min;
		result.intersection = add(pos, scale(dir, ts_min));
	}
	
	return result;
}

__host__ __device__ __host__ int lin_index(int x, int y, int w, int h) {
	return (h - 1 - y) * w + x;
}

__global__ void antiaaliasing(uchar4 *data, uchar4 *odata, int w, int h, int r) {
	int idx     = blockDim.x * blockIdx.x + threadIdx.x;
	int idy     = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;

	//~ int r = 2;
	for (int x = idx; x < w; x += offsetx)
		for (int y = idy; y < h; y += offsety) {
			int vx = 0, vy = 0, vz = 0;

			for (int dx = -r; dx <= r; dx++)
				for (int dy = -r; dy <= r; dy++) {
					int fx = x + dx;
					int fy = y + dy;

					if (fx >= 0 && fx < w && fy >= 0 && fy < h) {
						vx += data[lin_index(fx, fy, w, h)].x;
						vy += data[lin_index(fx, fy, w, h)].y;
						vz += data[lin_index(fx, fy, w, h)].z;
					}
				}
			odata[lin_index(x, y, w, h)].x = vx / ((r+1) * (r+1) * 4.0);
			odata[lin_index(x, y, w, h)].y = vy / ((r+1) * (r+1) * 4.0);
			odata[lin_index(x, y, w, h)].z = vz / ((r+1) * (r+1) * 4.0);
			//~ data[lin_index(x, y, w, h)].x /= 2;
			//~ data[lin_index(x, y, w, h)].y /= 2;
			//~ data[lin_index(x, y, w, h)].z /= 2;
		}
}


__host__ __device__ vect3 calc_light(
	Triangle *triags,
	int trig_cnt,
	ray_intersection pseen,
	Light light,
	vect3 pc
) {
	vect3 dir_lit = norm(diff(pseen.intersection, light.pos));
	ray_intersection plit;
	double sc = 0;
	vect3 col_sc = {sc, sc, sc};

	plit = ray(triags, trig_cnt, light.pos, dir_lit);

	/*
	if (plit.index == pseen.index) {
		double light_distance = plit.dst_min / light.intensity;
		
		if (light_distance < 1)
			light_distance = 1;
		sc = dot(triags[pseen.index].normal, dir_lit) / light_distance;
		
		sc = abs(sc);

		col_sc.x = light.col.x * sc;
		col_sc.y = light.col.y * sc;
		col_sc.z = light.col.z * sc;

		col_sc.x = (1 - ambient) * col_sc.x + ambient;
		col_sc.y = (1 - ambient) * col_sc.y + ambient;
		col_sc.z = (1 - ambient) * col_sc.z + ambient;
	}
	*/
	if (plit.index == pseen.index) {
		double distance_fade = (light.intensity - plit.dst_min) / light.intensity;
		if (distance_fade > 0) {
			sc = distance_fade * dot(triags[pseen.index].normal, dir_lit);
			
			sc = abs(sc);

			col_sc.x = light.col.x * sc;
			col_sc.y = light.col.y * sc;
			col_sc.z = light.col.z * sc;

			vect3 dir2lit = norm(diff(light.pos, pseen.intersection));
			vect3 dir2cam = norm(diff(pc,        pseen.intersection));
			
			double a_fall = dot(triags[pseen.index].normal, dir2lit);
			double a_refl = dot(triags[pseen.index].normal, dir2cam);
			double a_diff = abs(a_fall - a_refl);

			vect3 n1 = prod( prod(triags[pseen.index].normal, dir2lit), dir2cam);
			//~ vect3 n2 = norm(prod(triags[pseen.index].normal, dir2cam));
			double l_diff = length(n1);

			//~ double full_reflcoef = distance_fade * pow( (1 - l_diff) * (1 - a_diff), 2 );
			//~ if (l_diff * a_diff < 0.0001) 
			//~ col_sc = add(col_sc, scale(light.col, full_reflcoef));
			
		}
	}
	
	return col_sc;
}

__host__ __device__ uchar4 raytrace_color(
	Triangle *triags, int trig_cnt,
	Light *light, int light_cnt,
	vect3 ambient,
	vect3 pc, vect3 dir_seen
) {
	uchar4 col = {0, 0, 0, 0};
	ray_intersection pseen = ray(triags, trig_cnt, pc, dir_seen);
	
	if (pseen.index >= 0) {
		vect3 sum_sc = {0, 0, 0};

		for (int lind = 0; lind < light_cnt; ++lind) {
			//~ sum_sc += calc_light(triags, trig_cnt, pseen, light[lnum]);
			sum_sc = add(
				sum_sc,
				calc_light(triags, trig_cnt, pseen, light[lind], pc)
			);

			//~ vect3 dir_lit = norm(diff(pseen.intersection, light[lind].pos));
			//~ double a_fall = dot(dir_lit,  triags[pseen.index].normal);
			//~ double a_refl = dot(dir_seen, triags[pseen.index].normal);

			//~ if (abs(abs(a_fall) - abs(a_refl)) < 0.1)
				//~ sum_sc = add(sum_sc, {1.0, 1.0, 1.0});
		}

		sum_sc.x = MIN(sum_sc.x, 1.0);
		sum_sc.y = MIN(sum_sc.y, 1.0);
		sum_sc.z = MIN(sum_sc.z, 1.0);
		
		//~ sum_sc = v2limits(sum_sc, ambient, {0.8, 0.8, 0.8});
		sum_sc = v2limits(sum_sc, ambient, {1, 1, 1});
		
		col.x = sum_sc.x * triags[pseen.index].color.x;
		col.y = sum_sc.y * triags[pseen.index].color.y;
		col.z = sum_sc.z * triags[pseen.index].color.z;
	}

#ifdef DEBUG
	for (int lind = 0; lind < light_cnt; ++lind) {
		vect3 lightvect = diff(light[lind].pos, pc);
		double lightcos = abs(dot(dir_seen, norm(lightvect)));
		if (lightcos > 0.99) {
			col.x = light[lind].col.x * 255;
			col.y = light[lind].col.y * 255;
			col.z = light[lind].col.z * 255;
			break;
		}
	}
#endif
	return col;
}

__global__ void raytrace(
	Triangle *triags, int trig_cnt,
	Light *light, int light_cnt,
	vect3 ambient,
	vect3 pc, vect3 pv,
	int w, int h,
	double angle, uchar4 *data
) {
	int idx     = blockDim.x * blockIdx.x + threadIdx.x;
	int idy     = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	
	double dw = 2.0 / (w - 1.0);
	double dh = 2.0 / (h - 1.0);
	double z  = 1.0 / tan(angle * M_PI / 360.0);
	
	vect3 bz = norm(diff(pv, pc));
	vect3 bx = norm(prod(bz, {0.0, 0.0, 1.0}));
	vect3 by = norm(prod(bx, bz));
	
	for (int x = idx; x < w; x += offsetx)	
		for (int y = idy; y < h; y += offsety) {
			double tx = -1.0 + dw * x;
			double ty = -1.0 + dh * y;
			
			vect3 v = {
				tx, 
				ty * h / w, 
				z
			}; 
			vect3 dir_seen = norm(mult(bx, by, bz, v)); 
			data[lin_index(x, y, w, h)] = raytrace_color(triags, trig_cnt, light, light_cnt, ambient, pc, dir_seen);
		}
}


void cpu(
	Triangle *triags, int trig_cnt,
	Light *light, int light_cnt,
	vect3 ambient,
	vect3 pc, vect3 pv,
	int w, int h,
	double angle, uchar4 *data
) {

	double dw = 2.0 / (w - 1.0);
	double dh = 2.0 / (h - 1.0);
	double z  = 1.0 / tan(angle * M_PI / 360.0);
	
	vect3 bz = norm(diff(pv, pc));
	vect3 bx = norm(prod(bz, {0.0, 0.0, 1.0}));
	vect3 by = norm(prod(bx, bz));
	
	int x, y;
	#pragma omp parallel for private(x, y) shared(data, bx, by, bz, triags, trig_cnt, pc, light, light_cnt, ambient)
	for (x = 0; x < w; x += 1)	
		for (y = 0; y < h; y += 1) {
			double tx = -1.0 + dw * x;
			double ty = -1.0 + dh * y;
			
			vect3 v = {
				tx, 
				ty * h / w, 
				z
			}; 
			vect3 dir_seen = norm(mult(bx, by, bz, v)); 
			data[lin_index(x, y, w, h)] = raytrace_color(triags, trig_cnt, light, light_cnt, ambient, pc, dir_seen);
		}
}
