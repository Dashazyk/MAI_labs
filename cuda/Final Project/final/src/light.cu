#include "hip/hip_runtime.h"
#include <math.h>
#include "light.h"

Light make_light(vect3 pos, double intensity) {
	return make_light(pos, {1, 1, 1}, intensity);
}

Light make_light(vect3 pos, vect3 col, double intensity) {
	return {
		pos,
		col,
		intensity
	};
}

void gen_light_circle(Light *lights, int light_cnt, double rad, double z, double intensity) {
	int i;
	double dphi = M_PI * 2 / light_cnt;

	for (i = 0; i < light_cnt; ++i) {
		double lx = rad * cos(dphi * i);
		double ly = rad * sin(dphi * i);

		lights[i] = make_light(
			{lx, ly, z},
			//~ {cos(dphi * i)/2+0.5, sin(dphi * i)/2+0.5, 0.5},
			{(double)((i%3)==0), (double)((i%3)==1), (double)((i%3)==2)},
			intensity
		);
	}
}
