#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h> 
#include <math.h>
#include <iostream>
#include "Triangle.h"
#include "light.h"
#include "vect3.h"
#include "render.h"
#include "fig.h"

#ifndef GRID_SIZE
#define GRID_SIZE 16
#endif 

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 16
#endif 

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)


struct cam {
    double r, z, phi;
    double Ar, Az;
    double omega_r, omega_z, omega_phi;
    double pr, pz;
};

using namespace std;

double get_ms() {
    struct timespec _t;
    clock_gettime(CLOCK_REALTIME, &_t);
    return _t.tv_sec*1000 + (_t.tv_nsec/1.0e6);
}

int main(int argc, char **argv) {
	int k, w, h;
	char buff[256];
	uchar4 *data;
	vect3 pc, pv;
	Light *lights;
	int fcnt = 5;
	int lcnt = 1;
	int tcnt = 0;
	Figure   *figs = (Figure *)malloc(sizeof(Figure) * fcnt);
	Light    *gpu_light;
	Triangle *gpu_triags;
	uchar4   *gpu_data;
	uchar4   *gpu_aata;
	int frames = 126;
	char output_name[256];
	double angle = 120.0;
	cam mc, mn;
	
	lights = (Light *)malloc(sizeof(Light) * lcnt);
	lights[0] = make_light({10, 1, 10}, 200);

	vect3  plane_color;
	vect3  plane_pos[4];

	bool is_gpu = true;
	bool is_sane = true;

	if (argc == 2 && !strcmp("--default", argv[1])) {
		printf("200");
		printf("800 600");
		printf("res/%%04d.data");
		printf("60");
		printf("8.0 3.0 0.0 2.0 1.0 2.0 6.0 1.0 0.0 0.0");
		printf("1.0 0.0 0.0 0.5 0.1 1.0 4.0 1.0 0.0 0.0");
		printf("-300  300 0");
		printf(" 300 -300 0");
		printf("-300 -300 0");
		printf(" 300  300 0");
		printf("1 0.9 0.9");
		printf( "1.0  1.0 1.0 0.5 0.0 0.5 0.3");
		printf("-1.0 -1.0 1.0 0.5 0.0 0.5 0.0");
		printf("1.0 -1.0 1.0 0.5 0.5 0.7 0.0");
		
		return 0;
	}
	else if (argc == 2 && !strcmp("--stb", argv[1]))
		is_sane = true;
	else if (argc == 2 && !strcmp("--cpu", argv[1]))
		is_gpu = false;
	
	vect3  dodec_color;
	vect3  dodec_pos;
	double dodec_rad;
	vect3  hexae_color;
	vect3  hexae_pos;
	double hexae_rad;
	vect3  icosa_color;
	vect3  icosa_pos;
	double icosa_rad;

	cin >> frames;
	cin >> w >> h;
	data = (uchar4*)malloc(sizeof(uchar4) * w * h);
	cin >> output_name;
	cin >> angle;

	cin >> mc.r >> mc.z >> mc.phi;
    cin >> mc.Ar >> mc.Az;
    cin >> mc.omega_r >> mc.omega_z >> mc.omega_phi;
    cin >> mc.pr >> mc.pz;
    
    cin >> mn.r >> mn.z >> mn.phi;
    cin >> mn.Ar >> mn.Az;
    cin >> mn.omega_r >> mn.omega_z >> mn.omega_phi;
    cin >> mn.pr >> mn.pz;
    
    cin >> plane_pos[0].x >> plane_pos[0].y >> plane_pos[0].z;
    cin >> plane_pos[1].x >> plane_pos[1].y >> plane_pos[1].z;
    cin >> plane_pos[2].x >> plane_pos[2].y >> plane_pos[2].z;
    cin >> plane_pos[3].x >> plane_pos[3].y >> plane_pos[3].z;
    cin >> plane_color.x >> plane_color.y >> plane_color.z;

	cin >> dodec_pos.x >> dodec_pos.y >> dodec_pos.z;
	cin >> dodec_rad;
	cin >> dodec_color.x >> dodec_color.y >> dodec_color.z;

	cin >> hexae_pos.x >> hexae_pos.y >> hexae_pos.z;
	cin >> hexae_rad;
	cin >> hexae_color.x >> hexae_color.y >> hexae_color.z;
	
	cin >> icosa_pos.x >> icosa_pos.y >> icosa_pos.z;
	cin >> icosa_rad;
	cin >> icosa_color.x >> icosa_color.y >> icosa_color.z;

	figs[0] = gen_floor(plane_color, plane_pos);
	figs[1] = gen_icosa(icosa_color, icosa_pos, icosa_rad);
	figs[2] = gen_hex  (hexae_color, hexae_pos, hexae_rad);
	figs[3] = gen_dodec(dodec_color, dodec_pos, dodec_rad);
	figs[4] = gen_dodec(dodec_color, {lights[0].pos.x, lights[0].pos.y, 0.15}, 0.3);

	for (int i = 0; i < fcnt; ++i)
		tcnt += figs[i].triag_count;

	printf("sane output: %d\n", is_sane);
	if (is_gpu) {
		CSC(hipMalloc(&gpu_triags, sizeof(Triangle) * tcnt));
		CSC(hipMalloc(&gpu_light,  sizeof(Light)    * lcnt));
		CSC(hipMalloc(&gpu_data,   sizeof(uchar4)   * w * h));
		CSC(hipMalloc(&gpu_aata,   sizeof(uchar4)   * w * h));

		CSC(hipMemcpy(gpu_light, lights, sizeof(Light) * lcnt, hipMemcpyHostToDevice));

		int tmp = 0;
		for (int i = 0; i < fcnt; ++i) {
			CSC(hipMemcpy(gpu_triags + tmp, figs[i].triags, sizeof(Triangle) * figs[i].triag_count, hipMemcpyHostToDevice));
			tmp += figs[i].triag_count;
		}
	}

	for (k = 0; k < frames; k++) {
		double t = 2 * M_PI * k / frames;
        
        pc = (vect3) {
            mc.r + mc.Ar * sin(mc.omega_r * t + mc.pr),
            mc.z + mc.Az * sin(mc.omega_z * t + mc.pz),
            mc.phi + mc.omega_phi * t
        };
        
        pv = (vect3) {
            mn.r + mn.Ar * sin(mn.omega_r * t + mn.pr),
            mn.z + mn.Az * sin(mn.omega_z * t + mn.pz),
            mn.phi + mn.omega_phi * t
        };
        
        pc = (vect3) {pc.x * cos(pc.z), pc.x * sin(pc.z), pc.y};
        pv = (vect3) {pv.x * cos(pv.z), pv.x * sin(pv.z), pv.y};

		if (is_gpu) {
			raytrace<<<dim3(GRID_SIZE, GRID_SIZE), dim3(BLOCK_SIZE, BLOCK_SIZE)>>> (
				gpu_triags, tcnt,
				gpu_light,  lcnt,
				{0.15, 0.15, 0.15}, // ambient
				pc, pv,
				w, h, angle,
				gpu_data
			);
			CSC(hipGetLastError ());
			antiaaliasing<<<dim3(GRID_SIZE, GRID_SIZE), dim3(BLOCK_SIZE, BLOCK_SIZE)>>> (gpu_data, gpu_aata, w, h, 1);
			CSC(hipGetLastError ());
			
			CSC(hipMemcpy(data, gpu_aata, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));
		}
		else {
			//~ Triangle *triags = (Triangle *)malloc(sizeof(Triangle) * tcnt);
			//~ tmp = 0;
			//~ memcpy(triags + tmp, floor.triags, sizeof(Triangle) * floor.triag_count);
			//~ tmp += floor.triag_count;
			//~ memcpy(triags + tmp, icosa.triags, sizeof(Triangle) * icosa.triag_count);
			//~ tmp += icosa.triag_count;
			//~ memcpy(triags + tmp, hexae.triags, sizeof(Triangle) * hexae.triag_count);
			//~ tmp += hexae.triag_count;
			//~ memcpy(triags + tmp, dodec.triags, sizeof(Triangle) * dodec.triag_count);
			//~ tmp += dodec.triag_count;

			//~ cpu(triags, tmp, lights[0], pc, pv, w, h, angle, data);
		}


		printf("%04d ", k);
		fflush(stdout);
		if (is_sane) {
			// based on convinient uchar4 layout in memory
			sprintf(buff, "res/%04d.jpg", k);
			printf("%s\n", buff);
			stbi_write_jpg(buff, w, h, 4, data, 100);
		}
		else {
			sprintf(buff, output_name, k);

			FILE *out = fopen(buff, "wb");
			fwrite(&w, sizeof(int), 1, out);
			fwrite(&h, sizeof(int), 1, out);	
			fwrite(data, sizeof(uchar4), w * h, out);
			fclose(out);
		}
	}
	printf("\n");
	free(data);	
	return 0;
}
