#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

// #define SIGN(x) ((x) > 0 ? 1 : ((x) ? -1 : 0))

#define CSC(call)  					\
	do {								\
		hipError_t res = call;			\
		if (res != hipSuccess) {		\
			fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
					__FILE__, __LINE__, hipGetErrorString(res));		\
			exit(0);					\
		}								\
	} while(0)

using namespace std;

#define co1 1.8
#define co2 2.8

__host__ __device__ double fun(double x) {
	return tan(co1 * x) - co2 * x;
}

__host__ __device__ double pi(void) {
	return M_PI / co1;
}

__device__ double bisection(double left, double right, double eps, int limit = 1000000) {
	double mid = 0;
	double fl  = fun(left);
	double fr  = fun(right);

	while (right - left > eps && abs(right - left) > abs(eps) && limit) {
		double fm;

		limit--;

		mid = (left + right) / 2;

		fm = fun(mid);
		// cout << left << "' '" << mid << " " << right << endl;
		// cout << fl   << "   " << fm  << " " << fr    << endl;
		// cout << endl;

		if (fm) {
			// если одинак знак с левого края и по середине
			if (fm * fl > 0) {
				fl   = fm;
				left = mid;
			}
			// если одинак знак с правого края и по середине
			if (fm * fr > 0) {
				fr    = fm;
				right = mid;
			}
		}
		else
			break;
	}

	return mid;
}

__global__ 
void caller(double *borders, double *results, double eps) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	results[idx] = bisection(borders[idx] + eps, borders[idx + 1] - eps, eps);
}

int main(void) {
	double left, right, eps;
	cout << "l, r, e: ";
	cin  >> left >> right >> eps;
	hipEvent_t start, end;
	bool has_zero = false;

	if (eps > 0.0001)
		cout << "Epsilon is too big" << endl;

	// next, we kinda wanna to find all the tear points, don't we?
	double tear = floor(left / pi()) * pi() - pi() / 2;

	has_zero = left < 0 && right > 0;

	int size = (right + pi() - left) / pi() + 2;
	if (has_zero)
		size += 2;
		
	thrust::device_vector<double> tear_points(size);
	thrust::device_vector<double> results    (size - 1);

	thrust::sequence(tear_points.begin(), tear_points.end(), tear, pi());

	if (has_zero) {
		tear_points[size - 1] = 0 - 100 * eps;
		tear_points[size - 2] = 0 + 100 * eps;
		thrust::sort(tear_points.begin(), tear_points.end());
	}

	// we should add 0 as break point, otherwise there are 3 roots in there
	

	cout << "Size: "    << size << endl;
	cout << "Called the kernel" << endl;

	//~ for (int i = 0; i < tear_points.size(); ++i)
		//~ cout << tear_points[i] << endl;

	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&end  ));
	CSC(hipEventRecord( start));
	caller<<<1, size - 1>>>(
		thrust::raw_pointer_cast(tear_points.data()),
		thrust::raw_pointer_cast(results.data()), 
		eps
	);
	CSC(hipGetLastError());

	CSC(hipEventRecord     (end));
	CSC(hipEventSynchronize(end));
	float t;
	CSC(hipEventElapsedTime(&t, start, end));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(end));		

	cout << "Results: " << endl;	
	for (int i = 1; i < results.size(); ++i) {
        double m = results[i];
        if (m > left && m < right) {
			cout << "Interval: [" << tear_points[i] << ", " << tear_points[i + 1] << "]" << endl;
			cout << "fun(" << m << ") = " << fun(m) << "\t (" << (int)fun(m) << ")" << endl;
			cout << endl;
		}
    }
    cout << "Time: " << t << endl;
	return 0;
}


